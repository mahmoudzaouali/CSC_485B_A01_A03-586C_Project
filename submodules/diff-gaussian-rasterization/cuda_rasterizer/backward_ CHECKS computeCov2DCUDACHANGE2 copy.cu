#include "hip/hip_runtime.h"
// Backward version of INVERSE 2D covariance matrix computation
// (due to length launched as separate kernel before other 
// backward steps contained in preprocess)
__global__ void computeCov2DCUDA(int P,
	const float3* means,
	const int* radii,
	const float* cov3Ds,
	const float h_x, float h_y,
	const float tan_fovx, float tan_fovy,
	const float* view_matrix,
	const float* dL_dconics,
	float3* dL_dmeans,
	float* dL_dcov)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	// Reading location of 3D covariance for this Gaussian
	const float* cov3D = cov3Ds + 6 * idx;

	// Fetch gradients, recompute 2D covariance and relevant 
	// intermediate forward results needed in the backward.
	float3 mean = means[idx];
	float3 dL_dconic = { dL_dconics[4 * idx], dL_dconics[4 * idx + 1], dL_dconics[4 * idx + 3] };
	float3 t = transformPoint4x3(mean, view_matrix);
	
	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;
	
	const float x_grad_mul = txtz < -limx || txtz > limx ? 0 : 1;
	const float y_grad_mul = tytz < -limy || tytz > limy ? 0 : 1;

	glm::mat3 J = glm::mat3(h_x / t.z, 0.0f, -(h_x * t.x) / (t.z * t.z),
		0.0f, h_y / t.z, -(h_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		view_matrix[0], view_matrix[4], view_matrix[8],
		view_matrix[1], view_matrix[5], view_matrix[9],
		view_matrix[2], view_matrix[6], view_matrix[10]);

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 T = W * J;

	glm::mat3 cov2D = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Use helper variables for 2D covariance entries. More compact.
	float a = cov2D[0][0] += 0.3f;
	float b = cov2D[0][1];
	float c = cov2D[1][1] += 0.3f;

	float denom = a * c - b * b;
	float dL_da = 0, dL_db = 0, dL_dc = 0;
	float denom2inv = 1.0f / ((denom * denom) + 0.0000001f);

		// Gradients of loss w.r.t. entries of 2D covariance matrix,
		// given gradients of loss w.r.t. conic matrix (inverse covariance matrix).
		// e.g., dL / da = dL / d_conic_a * d_conic_a / d_a
		dL_da = denom2inv != 0 ? denom2inv * (-c * c * dL_dconic.x + 2 * b * c * dL_dconic.y + (denom - a * c) * dL_dconic.z) : 0;
		dL_dc = denom2inv != 0 ? denom2inv * (-a * a * dL_dconic.z + 2 * a * b * dL_dconic.y + (denom - a * c) * dL_dconic.x) : 0;
		dL_db = denom2inv != 0 ? denom2inv * 2 * (b * c * dL_dconic.x - (denom + 2 * b * b) * dL_dconic.y + a * b * dL_dconic.z) : 0;

		// Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry, 
		// given gradients w.r.t. 2D covariance matrix (diagonal).
		// cov2D = transpose(T) * transpose(Vrk) * T;
		dL_dcov[6 * idx + 0] =  denom2inv != 0 ? (T[0][0] * T[0][0] * dL_da + T[0][0] * T[1][0] * dL_db + T[1][0] * T[1][0] * dL_dc) : 0; //Set to 0 if denom2inv == 0
		dL_dcov[6 * idx + 3] = denom2inv != 0 ? (T[0][1] * T[0][1] * dL_da + T[0][1] * T[1][1] * dL_db + T[1][1] * T[1][1] * dL_dc) : 0; //Set to 0 if denom2inv == 0
		dL_dcov[6 * idx + 5] = denom2inv != 0 ? (T[0][2] * T[0][2] * dL_da + T[0][2] * T[1][2] * dL_db + T[1][2] * T[1][2] * dL_dc) : 0; //Set to 0 if denom2inv == 0

		// Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry, 
		// given gradients w.r.t. 2D covariance matrix (off-diagonal).
		// Off-diagonal elements appear twice --> double the gradient.
		// cov2D = transpose(T) * transpose(Vrk) * T;
		dL_dcov[6 * idx + 1] = denom2inv != 0 ? 2 * T[0][0] * T[0][1] * dL_da + (T[0][0] * T[1][1] + T[0][1] * T[1][0]) * dL_db + 2 * T[1][0] * T[1][1] * dL_dc : 0; //Set to 0 if denom2inv == 0
		dL_dcov[6 * idx + 2] = denom2inv != 0 ? 2 * T[0][0] * T[0][2] * dL_da + (T[0][0] * T[1][2] + T[0][2] * T[1][0]) * dL_db + 2 * T[1][0] * T[1][2] * dL_dc : 0; //Set to 0 if denom2inv == 0
		dL_dcov[6 * idx + 4] = denom2inv != 0 ? 2 * T[0][2] * T[0][1] * dL_da + (T[0][1] * T[1][2] + T[0][2] * T[1][1]) * dL_db + 2 * T[1][1] * T[1][2] * dL_dc : 0; //Set to 0 if denom2inv == 0

	// Gradients of loss w.r.t. upper 2x3 portion of intermediate matrix T
	// cov2D = transpose(T) * transpose(Vrk) * T;
	float dL_dT00 = 2 * (T[0][0] * Vrk[0][0] + T[0][1] * Vrk[0][1] + T[0][2] * Vrk[0][2]) * dL_da +
		(T[1][0] * Vrk[0][0] + T[1][1] * Vrk[0][1] + T[1][2] * Vrk[0][2]) * dL_db;
	float dL_dT01 = 2 * (T[0][0] * Vrk[1][0] + T[0][1] * Vrk[1][1] + T[0][2] * Vrk[1][2]) * dL_da +
		(T[1][0] * Vrk[1][0] + T[1][1] * Vrk[1][1] + T[1][2] * Vrk[1][2]) * dL_db;
	float dL_dT02 = 2 * (T[0][0] * Vrk[2][0] + T[0][1] * Vrk[2][1] + T[0][2] * Vrk[2][2]) * dL_da +
		(T[1][0] * Vrk[2][0] + T[1][1] * Vrk[2][1] + T[1][2] * Vrk[2][2]) * dL_db;
	float dL_dT10 = 2 * (T[1][0] * Vrk[0][0] + T[1][1] * Vrk[0][1] + T[1][2] * Vrk[0][2]) * dL_dc +
		(T[0][0] * Vrk[0][0] + T[0][1] * Vrk[0][1] + T[0][2] * Vrk[0][2]) * dL_db;
	float dL_dT11 = 2 * (T[1][0] * Vrk[1][0] + T[1][1] * Vrk[1][1] + T[1][2] * Vrk[1][2]) * dL_dc +
		(T[0][0] * Vrk[1][0] + T[0][1] * Vrk[1][1] + T[0][2] * Vrk[1][2]) * dL_db;
	float dL_dT12 = 2 * (T[1][0] * Vrk[2][0] + T[1][1] * Vrk[2][1] + T[1][2] * Vrk[2][2]) * dL_dc +
		(T[0][0] * Vrk[2][0] + T[0][1] * Vrk[2][1] + T[0][2] * Vrk[2][2]) * dL_db;

	// Gradients of loss w.r.t. upper 3x2 non-zero entries of Jacobian matrix
	// T = W * J
	float dL_dJ00 = W[0][0] * dL_dT00 + W[0][1] * dL_dT01 + W[0][2] * dL_dT02;
	float dL_dJ02 = W[2][0] * dL_dT00 + W[2][1] * dL_dT01 + W[2][2] * dL_dT02;
	float dL_dJ11 = W[1][0] * dL_dT10 + W[1][1] * dL_dT11 + W[1][2] * dL_dT12;
	float dL_dJ12 = W[2][0] * dL_dT10 + W[2][1] * dL_dT11 + W[2][2] * dL_dT12;

	float tz = 1.f / t.z;
	float tz2 = tz * tz;
	float tz3 = tz2 * tz;

	// Gradients of loss w.r.t. transformed Gaussian mean t
	float dL_dtx = x_grad_mul * -h_x * tz2 * dL_dJ02;
	float dL_dty = y_grad_mul * -h_y * tz2 * dL_dJ12;
	float dL_dtz = -h_x * tz2 * dL_dJ00 - h_y * tz2 * dL_dJ11 + (2 * h_x * t.x) * tz3 * dL_dJ02 + (2 * h_y * t.y) * tz3 * dL_dJ12;

	// Account for transformation of mean to t
	// t = transformPoint4x3(mean, view_matrix);
	float3 dL_dmean = transformVec4x3Transpose({ dL_dtx, dL_dty, dL_dtz }, view_matrix);

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the covariance matrix.
	// Additional mean gradient is accumulated in BACKWARD::preprocess.
	dL_dmeans[idx] = dL_dmean;
}